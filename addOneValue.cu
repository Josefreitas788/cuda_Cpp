#include "hip/hip_runtime.h"
#include <iostream>


using namespace std;

//kernel function
__global__ void add(int *a, int *b, int *c)
{

    *c = *a + *b;
    
}

// code to run on the CPU
int main(){

    hipDeviceReset();
    
    int a, b, c;
    int *device_a, *device_b, *device_c;
    int size = sizeof(int);

    hipMalloc((void**)&device_a, size);
    hipMalloc((void**)&device_b, size);
    hipMalloc((void**)&device_c, size);

    a = 10;
    b = 40;

    hipMemcpy(device_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(device_b, &b, size, hipMemcpyHostToDevice);

    add<<<1,1>>>(device_a, device_b, device_c);

    hipMemcpy(&c, device_c, size, hipMemcpyDeviceToHost);
    cout << "a + b = " << c << "\n";
    hipFree(device_a);
    hipFree(device_b);
    hipFree(device_c);
    
    return 0;
}
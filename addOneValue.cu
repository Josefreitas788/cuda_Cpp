#include "hip/hip_runtime.h"
#include <iostream>


using namespace std;

//kernel function
__global__ void add(int *a, int *b, int *c)
{

    *c = *a + *b;
    
}

// code to run on the CPU
int main(){
    hipError_t cudaStatus;
    hipDeviceReset();
    
    int a, b, c;
    int *device_a, *device_b, *device_c;
    int size = sizeof(int);

    cudaStatus = hipMalloc((void**)&device_a, size);
    if(cudaStatus != hipSuccess){
        cout << "hipMalloc device_a failed!";
        return 1;
    }

    cudaStatus = hipMalloc((void**)&device_b, size);
    if(cudaStatus != hipSuccess){
        cout << "hipMalloc device_b failed!";
        return 1;
    }
    
    cudaStatus =  hipMalloc((void**)&device_c, size);
    if(cudaStatus != hipSuccess){
        cout << "hipMalloc device_c failed!";
        return 1;
    }

    a = 10;
    b = 40;

    cudaStatus = hipMemcpy(device_a, &a, size, hipMemcpyHostToDevice);
    if(cudaStatus != hipSuccess){
        cout << "hipMemcpy device_a to a failed!";
        return 1;
    }

    cudaStatus = hipMemcpy(device_b, &b, size, hipMemcpyHostToDevice);
    if(cudaStatus != hipSuccess){
        cout << "hipMemcpy device_b to b failed!";
        return 1;
    }

    add<<<1,1>>>(device_a, device_b, device_c);

    cudaStatus = hipMemcpy(&c, device_c, size, hipMemcpyDeviceToHost);
    if(cudaStatus != hipSuccess){
        cout << "hipMemcpy device_c to c failed!";
        return 1;
    }
    cout << "a + b = " << c << "\n";
    
    hipFree(device_a);
    hipFree(device_b);
    hipFree(device_c);
    
    return 0;
}
#include<stdio.h>
#include "hip/hip_runtime.h"
// kernel
__global__ void meuKernel(){
    
    printf("Hello World da GPU!\n");

    
}

//codigo de CPU

int main(){
    printf("Hello World!\n");
    meuKernel <<<2,2>>>();
    /*É preciso colocar para que a CPU espere a GPU terminar,
    se eu não colocar o programa terminar sem dar tempo da GPU Imprimir os dados*/
    hipDeviceSynchronize();
    return 0;
}

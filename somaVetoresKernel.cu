#include "hip/hip_runtime.h"

#include <iostream>


using namespace std;

__global__ void SomaDeVetores(int *a, int *b, int *c) {
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main(){
    hipDeviceReset();
    hipError_t cudaStatus;


    int *device_a, *device_b, *device_c;
    int *host_a, *host_b, *host_c;

    int tamanhoDoVetor = 1024;
    int size = tamanhoDoVetor * sizeof(int);

    host_a = (int *)malloc(size);
    host_b = (int *)malloc(size);
    host_c = (int *)malloc(size);

    for(int i = 0; i< tamanhoDoVetor ; i++){
        host_a[i] = i;
        host_b[i] = i;
    }

    cudaStatus = hipMalloc((void **)&device_a, size);
    if(cudaStatus != hipSuccess){
        cout << "hipMalloc failed!" << "\n";
        return 1;
    }
    cudaStatus = hipMalloc((void **)&device_b, size);
    if(cudaStatus != hipSuccess){
        cout << "hipMalloc failed!" << "\n";
        return 1;
    }
    cudaStatus = hipMalloc((void **)&device_c, size);
    if(cudaStatus != hipSuccess){
        cout << "hipMalloc failed!" << "\n";
        return 1;
    }

    cudaStatus = hipMemcpy(device_a, host_a, size, hipMemcpyHostToDevice);
    if(cudaStatus != hipSuccess){
        cout << "hipMemcpy failed!" << "\n";
        return 1;
    }
    cudaStatus = hipMemcpy(device_b, host_b, size, hipMemcpyHostToDevice);
    if(cudaStatus != hipSuccess){
        cout << "hipMemcpy failed!" << "\n";
        return 1;
    }

    SomaDeVetores<<<1,tamanhoDoVetor>>>(device_a, device_b, device_c);
    hipDeviceSynchronize();

    cudaStatus = hipMemcpy(host_c, device_c, size, hipMemcpyDeviceToHost);
    if(cudaStatus != hipSuccess){
        cout << "hipMemcpy failed!" << "\n";
        return 1;
    }

    for(int i = 0; i< tamanhoDoVetor ; i++){
        cout << host_c[i]   << " ";
    }
    cout << "\n";





}
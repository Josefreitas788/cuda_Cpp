#include "hip/hip_runtime.h"

#include <iostream>

using namespace std;

__global__ void MultiplicaPorEscalar(int *a, int *b, int n)
{
    int i = threadIdx.x;
    b[i] = a[i]*n;    
}

int main()
{
    hipError_t CudaStatus;
    int *host_a, *host_b;
    int *device_a, *device_b;

    int arraySize = 256;

    int size = arraySize*sizeof(int);

    host_a = (int*)malloc(size);
    host_b = (int*)malloc(size);

    for(int i=0; i<arraySize; i++)
    {
        host_a[i] = i;
    }

    CudaStatus = hipMalloc((void**)&device_a, size);
    if(CudaStatus != hipSuccess)
    {
        cout << "Error al reservar memoria en el dispositivo" << endl;
        return 1;
    }

    CudaStatus = hipMalloc((void**)&device_b, size);
    if(CudaStatus != hipSuccess)
    {
        cout << "Error al reservar memoria en el dispositivo" << endl;
        return 1;
    }

    CudaStatus = hipMemcpy(device_a, host_a, size, hipMemcpyHostToDevice);
    if(CudaStatus != hipSuccess)
    {
        cout << "Error al copiar datos del host al dispositivo" << endl;
        return 1;
    }

    MultiplicaPorEscalar<<<1, arraySize>>>(device_a, device_b, 3);
    hipDeviceSynchronize();

    CudaStatus = hipMemcpy(host_b, device_b, size, hipMemcpyDeviceToHost);
    if(CudaStatus != hipSuccess)
    {
        cout << "Error al copiar datos del dispositivo al host" << endl;
        return 1;
    }

    for(int i=0; i<arraySize; i++)
    {
        cout << host_b[i]<< " ";
    }
    cout << "\n";


}
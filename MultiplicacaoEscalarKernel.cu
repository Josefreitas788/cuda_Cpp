#include "hip/hip_runtime.h"

#include <iostream>

using namespace std;

__global__ void MultiplicaPorEscalar(int *a, int *b, int n)
{
    int i = threadIdx.x;
    b[i] = a[i]*n;    
}

int main()
{
    hipError_t CudaStatus;
    int *host_a, *host_b;
    int *device_a, *device_b;

    int arraySize = 256;

    int size = arraySize*sizeof(int);

    host_a = (int*)malloc(size);
    host_b = (int*)malloc(size);

    for(int i=0; i<arraySize; i++)
    {
        host_a[i] = i;
    }

    CudaStatus = hipMalloc((void**)&device_a, size);
    if(CudaStatus != hipSuccess)
    {
        cout << "hipMalloc failed" << "\n";
        return 1;
    }

    CudaStatus = hipMalloc((void**)&device_b, size);
    if(CudaStatus != hipSuccess)
    {
        cout << "hipMalloc failed" << "\n";
        return 1;
    }

    CudaStatus = hipMemcpy(device_a, host_a, size, hipMemcpyHostToDevice);
    if(CudaStatus != hipSuccess)
    {
        cout << "hipMalloc failed" << "\n";
        return 1;
    }

    MultiplicaPorEscalar<<<1, arraySize>>>(device_a, device_b, 3);
    hipDeviceSynchronize();

    CudaStatus = hipMemcpy(host_b, device_b, size, hipMemcpyDeviceToHost);
    if(CudaStatus != hipSuccess)
    {
        cout << "hipMalloc failed" << "\n";
        return 1;
    }

    for(int i=0; i<arraySize; i++)
    {
        cout << host_b[i]<< " ";
    }
    cout << "\n";


}
